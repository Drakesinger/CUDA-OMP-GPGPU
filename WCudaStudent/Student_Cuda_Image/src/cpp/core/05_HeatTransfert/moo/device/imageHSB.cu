#include "hip/hip_runtime.h"
#include <iostream>

#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"
#include "IndiceTools.h"
#include "CalibreurF.h"
#include "ColorTools.h"

using std::cout;
using std::endl;

__global__ void heatImageHSB(float* ptrDevIn, uchar4* ptrDevOut,CalibreurF calibreur, int w, int h);

__global__ void heatImageHSB(float* ptrDevIn, uchar4* ptrDevOut,CalibreurF calibreur, int w, int h)
    {

    int tid = Indice2D::tid();
    int nbThread = Indice2D::nbThread();
    const int MAX = w*h;

    int s = tid;
    while(s < MAX)
	{
	ptrDevOut[s].w = 255;
	calibreur.calibrer(ptrDevIn[s]);
	ColorTools::HSB_TO_RVB(ptrDevIn[s], 1.0f, 1.0f, &ptrDevOut[s]);
	s += nbThread;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
