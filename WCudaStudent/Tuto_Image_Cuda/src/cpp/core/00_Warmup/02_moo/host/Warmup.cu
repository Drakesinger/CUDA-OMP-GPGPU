#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Warmup.h"
#include "Device.h"
#include "MathTools.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void warmup(uchar4* ptrDevPixels,int w, int h,float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Warmup::Warmup(int w, int h,float dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->dt=dt;

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t=0;

    //Outputs
    this->title="Warmup_Cuda";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);

    cout << endl<<"[Warmup]  dt =" << dt;
    }

Warmup::~Warmup()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void Warmup::process(uchar4* ptrDevPixels, int w, int h)
    {
    warmup<<<dg,db>>>(ptrDevPixels,w,h,t);
    }

/**
 * Override
 * Call periodicly by the API
 */
void Warmup::animationStep()
    {
    t+=dt;
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
float Warmup::getAnimationPara(void)
    {
    return t;
    }

/**
 * Override
 */
int Warmup::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int Warmup::getH(void)
    {
    return  h;
    }

/**
 * Override
 */
string Warmup::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

