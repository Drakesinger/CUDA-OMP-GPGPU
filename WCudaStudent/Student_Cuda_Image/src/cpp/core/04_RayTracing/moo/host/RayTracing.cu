#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "DomaineMath.h"
#include "RayTracing.h"
#include "Device.h"

#include <math.h>
#include <limits>

#define BORD 200
#define LENGTH 500
__constant__ Sphere TAB_DATA_CM[LENGTH];

using std::cout;
using std::endl;
using std::string;

extern __global__ void rayTracing(uchar4* ptrDevPixels, int w, int h, Sphere* ptrDevSpheres, DomaineMath domaineMath, int length, float t);

extern Sphere* instanciateSpheres(Sphere*,int);
extern void destructSpheres(Sphere*);

RayTracing::RayTracing(int w, int h, DomaineMath* domaineMath, float dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->ptrDomaineMathInit = domaineMath;
    this->t = 0;
    this->dt = dt;

    // Tools
    this->dg = dim3(16, 8, 1);
    this->db = dim3(32, 2, 1);

    // Outputs
    this->title = "RayTracing";

    ptrSpheres = new Sphere[LENGTH];

    for(int i = 0; i < LENGTH; ++i)
	{
	float3 center;
	center.x = randomMinMax(BORD, w-BORD);
	center.y = randomMinMax(BORD, h-BORD);
	center.z = randomMinMax(10, 2*w);

	ptrSpheres[i] = *(new Sphere(center, randomMinMax(20,80), randomMinMax(0, 1)));
	}

    ptrDevSpheres = instanciateSpheres(ptrSpheres, LENGTH);
    }

RayTracing::~RayTracing()
    {
    destructSpheres(ptrDevSpheres);
    delete[] ptrSpheres;
    }

/*-------------------------*\
 |*     Methode override    *|
 \*-------------------------*/

void RayTracing::process(uchar4* ptrDevPixels, int w, int h, const DomaineMath& domaineMath)
    {
    rayTracing<<<dg,db>>>(ptrDevPixels, w, h, ptrDevSpheres, domaineMath,LENGTH ,t);
    }

float RayTracing::getAnimationPara(void)
    {
    return this->t;
    }

void RayTracing::animationStep()
    {
    this->t += dt;
    }

 /*--------------*\
 |* get override *|
 \*--------------*/
DomaineMath* RayTracing::getDomaineMathInit(void)
    {
    return ptrDomaineMathInit;
    }
float RayTracing::getT(void)
    {
    return t;
    }

string RayTracing::getTitle(void)
    {
    return title;
    }
int RayTracing::getW(void)
    {
    return w;
    }
int RayTracing::getH(void)
    {
    return h;
    }

float RayTracing::randomMinMax(float min, float max)
    {
    return (max-min)*((float)(rand())/(float)RAND_MAX) + min;
    }

__host__ ConstantMemoryLink constantMemoryLink(void)
    {
    Sphere* ptrDevTabData;
    size_t sizeAll = LENGTH * sizeof(Sphere);
    HANDLE_ERROR(hipGetSymbolAddress((void **) &ptrDevTabData, TAB_DATA_CM));
    ConstantMemoryLink cmLink =
       {
	       (void**) ptrDevTabData, LENGTH, sizeAll
       };
    return cmLink;
}

__host__ Sphere* instanciateSpheres(Sphere* ptrSpheres, int n)
    {
    ConstantMemoryLink cmLink = constantMemoryLink();
    Sphere* ptrDevSpheres = (Sphere*)cmLink.ptrDevTab;
    size_t sizeALL = cmLink.sizeAll;

    HANDLE_ERROR(hipMemcpy(ptrDevSpheres, ptrSpheres, sizeALL, hipMemcpyHostToDevice));

    return ptrDevSpheres;
    }

__host__ void destructSpheres(Sphere* ptrDevSpheres)
    {
    HANDLE_ERROR(hipFree(ptrDevSpheres));
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
