#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "cudaTools.h"
#include "Device.h"
#include "DomaineMath.h"
#include "AdvancedMath.h"



/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void advanced(uchar4* ptrDevPixels,int w, int h,DomaineMath domaineMath,float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void advanced(uchar4* ptrDevPixels,int w, int h,DomaineMath domaineMath,float t)
    {
    AdvancedMath advancedMath;

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    uchar4 color;

    double x;
    double y;

    int pixelI;
    int pixelJ;

    int s = TID;
    while (s < WH)
    	{
    	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)

    	// (i,j) domaine ecran
    	// (x,y) domaine math
    	domaineMath.toXY(pixelI, pixelJ, &x, &y); //  (i,j) -> (x,y)

    	advancedMath.colorXY(&color,x, y,t); // update color

    	ptrDevPixels[s] = color;

    	s += NB_THREAD;
    	}
    }





/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

