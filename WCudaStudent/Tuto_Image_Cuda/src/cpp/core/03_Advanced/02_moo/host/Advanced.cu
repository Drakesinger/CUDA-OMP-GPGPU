#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Advanced.h"
#include "Device.h"
#include "MathTools.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void advanced(uchar4* ptrDevPixels,int w, int h,DomaineMath domaineMath,float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Advanced::Advanced(int w, int h, float dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->dt=dt;

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t = 0;
    ptrDomaineMathInit = new DomaineMath(0, 0, 10, 10);

    //Outputs
    this->title = "Advanced_Cuda";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);
    }

Advanced::~Advanced()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void Advanced::process(uchar4* ptrDevPixels, int w, int h, const DomaineMath& domaineMath)
    {
    advanced<<<dg,db>>>(ptrDevPixels,w,h,domaineMath,t);
    }

/**
 * Override
 * Call periodicly by the API
 */
void Advanced::animationStep()
    {
    t+=dt;
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/


/**
 * Override
 */
DomaineMath* Advanced::getDomaineMathInit(void)
    {
    return ptrDomaineMathInit;
    }

/**
 * Override
 */
float Advanced::getAnimationPara(void)
    {
    return t;
    }

/**
 * Override
 */
int Advanced::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int Advanced::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string Advanced::getTitle(void)
    {
    return title;
    }


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

