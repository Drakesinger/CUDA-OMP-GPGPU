#include "hip/hip_runtime.h"
#include <assert.h>

#include "Damier.h"
#include "Device.h"
#include "MathTools.h"

using cpu::IntervalF;


/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void damier(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Damier::Damier(int w, int h, float dt, int n) :
	variateurAnimation(IntervalF(0, 2 * PI), dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->n = n;

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t = 0;
    ptrDomaineMathInit=new DomaineMath(0,0,2*PI,2*PI);

    //Outputs
    this->title = "Damier_CUDA (Zoomable)";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);
    }

Damier::~Damier()
    {
   delete ptrDomaineMathInit;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void Damier::process(uchar4* ptrDevPixels, int w, int h, const DomaineMath& domaineMath)
    {
    damier<<<dg,db>>>(ptrDevPixels,w,h,domaineMath,n,t);
    }

/**
 * Override
 * Call periodicly by the API
 */
void Damier::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
    }


/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
DomaineMath* Damier::getDomaineMathInit(void)
    {
    return ptrDomaineMathInit;
    }

/**
 * Override
 */
float Damier::getAnimationPara(void)
    {
    return t;
    }

/**
 * Override
 */
int Damier::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int Damier::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string Damier::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

