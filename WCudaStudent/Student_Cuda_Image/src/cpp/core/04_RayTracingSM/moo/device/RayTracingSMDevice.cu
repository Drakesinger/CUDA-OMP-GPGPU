#include "hip/hip_runtime.h"
#include <iostream>

#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"
#include "IndiceTools.h"

#include "RayTracingSMMath.h"

using std::cout;
using std::endl;

__global__ void rayTracingSM(uchar4* ptrDevPixels, int w, int h, Sphere* ptrDevSpheres,int length, float t)
    {

    extern __shared__ Sphere tabSphereSM[];

    int tid = Indice2D::tid();
    int nbThread = Indice2D::nbThread();
    const int MAX = w*h;
    int s = threadIdx.x+blockDim.x*threadIdx.y;

    while(s< length)
	{
	tabSphereSM[s] = ptrDevSpheres[s];
	s += blockDim.x * blockDim.y;
	}
    __syncthreads();

    RayTracingSMMath rayTracingMath(ptrDevSpheres,length);

    int i,j;
    s = tid;


    while(s < MAX)
	{
	IndiceTools::toIJ(s,w,&i,&j);

	rayTracingMath.colorXY(j, i, t, &ptrDevPixels[s]);
	s += nbThread;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
