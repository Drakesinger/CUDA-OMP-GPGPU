#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include <iostream>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/


__host__ void addVecteur(void);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __global__ void addVecteur(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n);
static __device__ void secondaire(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/


/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ void addVecteur(void)
    {
    int n=9;
    float* ptrV1 = new float[n];
    float* ptrV2 = new float[n];
    float* ptrW = new float[n];

    for(int i=0; i<n; i++)
	{
	ptrV1[i]=i+1;
	ptrV2[i]=(i+1)*10;
	}

    float* ptrDevV1= NULL;
    float* ptrDevV2= NULL;
    float* ptrDevW=NULL;
    size_t size=n*sizeof(float);

    HANDLE_ERROR(hipMalloc(&ptrDevV1, size));
    HANDLE_ERROR(hipMalloc(&ptrDevV2, size));
    HANDLE_ERROR(hipMalloc(&ptrDevW, size));

    HANDLE_ERROR(hipMemset(ptrDevW, 0, size));

    HANDLE_ERROR(hipMemcpy(ptrDevV1,ptrV1, size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(ptrDevV2,ptrV2, size, hipMemcpyHostToDevice));

    dim3 dg(16,2,1);
    dim3 db(32,4,1);

    addVecteur<<<dg,db>>>(ptrDevV1, ptrDevV2, ptrDevW, n);

    HANDLE_ERROR(hipMemcpy(ptrW,ptrDevW, size, hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(ptrDevV1));
    HANDLE_ERROR(hipFree(ptrDevV2));
    HANDLE_ERROR(hipFree(ptrDevW));

    cout << endl;
    for(int i=0; i<n;i++)
	{
	cout << ptrW[i] << "\t";
	}
    cout << endl;
    delete[] ptrV1;
    delete[] ptrV2;
    delete[] ptrW;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void addVecteur(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n)
    {
    secondaire(ptrDevV1, ptrDevV2, ptrDevW, n);
    }

__device__ void secondaire(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n)
    {
    const int NB_THREAD=Indice2D::nbThread();
    const int TID=Indice2D::tid();

    int s=TID;
    while(s<n)
	{
	ptrDevW[s]=ptrDevV1[s]+ptrDevV2[s];
	s+=NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

