#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "DomaineMath.h"
#include "Julia.h"
#include "Device.h"

using cpu::IntervalI;


using std::cout;
using std::endl;

extern __global__ void julia(uchar4* ptrDevPixels, int w, int h, DomaineMath ptrDomaineMathInit, int n, int nMax, float cx, float cy);

Julia::Julia(int w, int h, DomaineMath* domaineMath, int nMin, int nMax, float dt, float cx, float cy) :
	w(w),h(h),n(nMin),nMin(nMin),nMax(nMax),cx(cx),cy(cy),
	ptrDomaineMathInit(domaineMath),
	variateurAnimation(IntervalI(nMin, nMax), 1)
    {
// Tools
    this->dg = dim3(8, 8, 1);
    this->db = dim3(16, 16, 1);

// Outputs
    this->title = "Julia";
    }

Julia::~Julia()
    {
    // rien
    }

void Julia::animationStep()
    {
    this->n = variateurAnimation.varierAndGet();
    }

void Julia::process(uchar4* ptrDevPixels,int w, int h,const DomaineMath& domaineMath)
    {
    julia<<<dg,db>>>(ptrDevPixels, w, h, domaineMath, n, nMin, cx, cy);
    }

float Julia::getAnimationPara(void)
    {
    return variateurAnimation.get();
    }

DomaineMath* Julia::getDomaineMathInit(void)
{
return ptrDomaineMathInit;
}

float Julia::getT(void)
{
return n;
}

string Julia::getTitle(void)
{
return title;
}

int Julia::getW(void)
{
return w;
}

int Julia::getH(void)
{
return h;
}

