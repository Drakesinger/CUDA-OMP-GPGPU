#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>

#include "Indice2D.h"
#include "IndiceTools.h"
#include "cudaTools.h"


#include "WarmupMath.h"


/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void warmup(uchar4* ptrDevPixels,int w, int h,float t);


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/



/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void warmup(uchar4* ptrDevPixels, int w, int h, float t)
    {
    WarmupMath warmupMath = WarmupMath(w, h);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    uchar4 color;

    int pixelI;
    int pixelJ;

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)

	warmupMath.colorIJ(&color,pixelI, pixelJ, t); 	// update color
	ptrDevPixels[s] = color;

	s += NB_THREAD;
	}
    }


/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

