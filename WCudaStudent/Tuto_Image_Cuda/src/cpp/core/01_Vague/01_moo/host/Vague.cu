#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Vague.h"
#include "Device.h"
#include "MathTools.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void vague(uchar4* ptrDevPixels,int w, int h,float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Vague::Vague(int w, int h,float dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->dt=dt;

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t=0;

    //Outputs
    this->title="Vague_Cuda";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);

    cout << endl<<"[Vague]  dt =" << dt;
    }

Vague::~Vague()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void Vague::process(uchar4* ptrDevPixels, int w, int h)
    {
    vague<<<dg,db>>>(ptrDevPixels,w,h,t);
    }

/**
 * Override
 * Call periodicly by the API
 */
void Vague::animationStep()
    {
    t+=dt;
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
float Vague::getAnimationPara(void)
    {
    return t;
    }

/**
 * Override
 */
int Vague::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int Vague::getH(void)
    {
    return  h;
    }

/**
 * Override
 */
string Vague::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

